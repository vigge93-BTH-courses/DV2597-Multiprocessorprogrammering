/***************************************************************************
 *
 * Paralell version of Gauss-Jordan row reduction
 *
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#define MAX_SIZE 4096

typedef double matrix[MAX_SIZE][MAX_SIZE];

int	N;		/* matrix size */
int	maxnum;		/* max number of element*/
char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch */
matrix	A;		/* matrix A	*/
double	b[MAX_SIZE];	/* vector b */
double	y[MAX_SIZE];	/* vector y */

/* forward declarations */
void work(void);
void Init_Matrix(void);
void Print_Matrix(void);
void Init_Default(void);
int Read_Options(int, char**);

int
main(int argc, char** argv)
{
    srand(1);
    printf("Gauss Jordan\n");
    // int i, timestart, timeend, iter;

    Init_Default();		/* Init default values	*/
    Read_Options(argc, argv);	/* Read arguments	*/
    Init_Matrix();		/* Init the matrix	*/
    auto start = std::chrono::steady_clock::now();
    work();
    auto end = std::chrono::steady_clock::now();
    std::cout << "Elapsed time =  " << std::chrono::duration<double>(end - start).count() << " sec\n";
    if (PRINT == 1)
        Print_Matrix();
}

__device__
int getIndex(int y, int x) {
    return x + y*MAX_SIZE;
}

__global__
void division_kernel(double *A_d, double *b_d, double *y_d, int N, int k) {
    int t_idx = threadIdx.x + blockDim.x*blockIdx.x + k+1; // start thread index at k+1 since all indexes <= k is already eliminated.
    y_d[k] = b_d[k] / A_d[getIndex(k, k)];
    if (t_idx >= N) return; // Guard clause
    A_d[getIndex(k, t_idx)] = A_d[getIndex(k, t_idx)] / A_d[getIndex(k, k)];; /* Division step */
}

__global__
void elimination_kernel(double *A_d, double *b_d, double *y_d, int N, int k) {
    int t_idx = threadIdx.x + blockDim.x*blockIdx.x; // col
    int t_idy = threadIdx.y + blockDim.y*blockIdx.y; // row
    A_d[getIndex(k, k)] = 1.0;
    
    if (t_idy == k || t_idy >= N) return; // Guard clauses

    if (t_idx == 0) {
        double* ptr = t_idy > k ? &b_d[t_idy] : &y_d[t_idy];
        *ptr = *ptr - A_d[getIndex(t_idy, k)] * y_d[k];
        A_d[getIndex(t_idy, k)] = 0.0;
    }
    if (t_idx <= k || t_idx >= N) return;
    A_d[getIndex(t_idy, t_idx)] = A_d[getIndex(t_idy, t_idx)] - A_d[getIndex(t_idy, k)] * A_d[getIndex(k, t_idx)]; /* Elimination step */    
}

void
work(void)
{
    /* Gaussian elimination algorithm, Algo 8.4 from Grama */
    int divBlocks = 2;
    int threads_per_blockdiv = 2;
    dim3 elBlocks(2, 2, 1);
    dim3 threads_per_blockel = (2, 2, 1);
    double *A_d;
    double *b_d, *y_d;
    hipMalloc((void**)&A_d,MAX_SIZE*MAX_SIZE*sizeof(double));
    hipMalloc((void**)&b_d,MAX_SIZE*sizeof(double));
    hipMalloc((void**)&y_d,MAX_SIZE*sizeof(double));
    for (int i = 0; i < MAX_SIZE; i++) {
        hipMemcpy(&A_d[i*MAX_SIZE], A[i], MAX_SIZE*sizeof(double), hipMemcpyHostToDevice);
    }
    hipMemcpy(b_d, b, MAX_SIZE*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, MAX_SIZE*sizeof(double), hipMemcpyHostToDevice);
    // auto start = std::chrono::steady_clock::now();
    // auto end = std::chrono::steady_clock::now();
    for (int k = 0; k < N; k++) { /* Outer loop */
        // start = std::chrono::steady_clock::now();
        division_kernel<<<divBlocks, threads_per_blockdiv>>>(A_d, b_d, y_d, N, k);
        hipDeviceSynchronize();
        // printf("Error: %s\n", cudaGetErrorString(cudaGetLastError()));
        // end = std::chrono::steady_clock::now();
        // std::cout << "Division elapsed time =  " << std::chrono::duration<double>(end - start).count() << " sec\n";

        // start = std::chrono::steady_clock::now();
        elimination_kernel<<<elBlocks, threads_per_blockel>>>(A_d, b_d, y_d, N, k);
        hipDeviceSynchronize();
        // printf("Error: %s\n", cudaGetErrorString(cudaGetLastError()));
        // end = std::chrono::steady_clock::now();
        // std::cout << "Elimination elapsed time =  " << std::chrono::duration<double>(end - start).count() << " sec\n";
    }
    // for (int i = 0; i < MAX_SIZE; i++) {
    //     cudaMemcpy(A[i], &A_d[i*MAX_SIZE], MAX_SIZE*sizeof(double), cudaMemcpyDeviceToHost);
    // }
    hipMemcpy(b, b_d, MAX_SIZE*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y, y_d, MAX_SIZE*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(b_d);
    hipFree(y_d);
}

void
Init_Matrix()
{
    int i, j;

    printf("\nsize      = %dx%d ", N, N);
    printf("\nmaxnum    = %d \n", maxnum);
    printf("Init	  = %s \n", Init);
    printf("Initializing matrix...");

    if (strcmp(Init, "rand") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = (double)(rand() % maxnum) + 5.0;
                else
                    A[i][j] = (double)(rand() % maxnum) + 1.0;
            }
        }
    }
    if (strcmp(Init, "fast") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = 5.0;
                else
                    A[i][j] = 2.0;
            }
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }

    printf("done \n\n");
    if (PRINT == 1)
        Print_Matrix();
}

void
Print_Matrix()
{
    int i, j;

    // printf("Matrix A:\n");
    // for (i = 0; i < N; i++) {
    //     printf("[");
    //     for (j = 0; j < N; j++)
    //         printf(" %5.2f,", A[i][j]);
    //     printf("]\n");
    // }
    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", y[j]);
    printf("]\n");
    printf("\n\n");
}

void
Init_Default()
{
    N = 2048;
    Init = "fast";
    maxnum = 15.0;
    PRINT = 0;
}

int
Read_Options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-I init_type] fast/rand \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          Init      = rand");
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'I':
                --argc;
                Init = *++argv;
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
}