#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <iostream>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>

__global__
void swap(int* a, int* b) {
    *a ^= *b;
    *b ^= *a;
    *a ^= *b;
}

__global__
void oddEvenSort_kernel(int* numbers_d, int n, int stride) {
    int t_idx = threadIdx.x + blockDim.x*blockId.x;

    if (t_idx*stride >= n)
    for (int i = 1; i <= s; i++) {
        int idx = t_idx*stride;
        // if (i % 2 == 1) idx += 1;
        idx += i % 2 == 1 ? 1 : 0;
        for (int j = idx; j < idx + stride; j = j + 2) {
            if (numbers_d[j] > numbers_d[j + 1]) {
                swap(&numbers_d[j], &numbers_d[j + 1]);
            }
        }
        __syncthreads()
    }
}

// The odd-even sort algorithm
// Total number of odd phases + even phases = the number of elements to sort
void oddeven_sort(std::vector<int>& numbers)
{
    auto s = numbers.size();
    int num_blocks = 1;
    int threads_per_block = 2048;

    int* numbers_d;
    int stride = (int)std::ceil(s/2048);
    auto s_bytes = s*sizeof(int)
    hipMalloc((void**)&numbers_d, s_bytes);
    hipMemcpy(numbers_d, numbers, s_bytes, hipMemcpyHostToDevice);

    oddeven_sort_kernel<<<num_blocks, threads_per_block>>>(numbers_d, n, stride);

    hipFree(numbers_d);
}

void print_sort_status(std::vector<int> numbers)
{
    std::cout << "The input is sorted?: " << (std::is_sorted(numbers.begin(), numbers.end()) == 0 ? "False" : "True") << std::endl;
}

int main()
{
    constexpr unsigned int size = 100'000; // Number of elements in the input

    // Initialize a vector with integers of value 0
    std::vector<int> numbers(size);
    // Populate our vector with (pseudo)random numbers
    srand(time(0));
    std::generate(numbers.begin(), numbers.end(), rand);

    print_sort_status(numbers);
    auto start = std::chrono::steady_clock::now();
    oddeven_sort(numbers);
    auto end = std::chrono::steady_clock::now();
    print_sort_status(numbers);
    std::cout << "Elapsed time =  " << std::chrono::duration<double>(end - start).count() << " sec\n";
}